#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>

#include "LaunchParams.h"

using namespace osc;

namespace osc {
  
  /*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
  extern "C" __constant__ LaunchParams optixLaunchParams;

  // for this simple example, we have a single ray type
  enum { SURFACE_RAY_TYPE=0, RAY_TYPE_COUNT };
  
  static __forceinline__ __device__
  void *unpackPointer( uint32_t i0, uint32_t i1 )
  {
    const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr ); 
    return ptr;
  }

  static __forceinline__ __device__
  void  packPointer( void* ptr01,
                      void* ptr23, 
                      void* ptr45, 
                      void* ptr67, 
                      uint32_t& i0, uint32_t& i1, 
                      uint32_t& i2, uint32_t& i3, 
                      uint32_t& i4, uint32_t& i5,
                      uint32_t& i6, uint32_t& i7)
  {
    const uint64_t uptr1 = reinterpret_cast<uint64_t>( ptr01 );
    i0 = uptr1 >> 32;
    i1 = uptr1 & 0x00000000ffffffff;
    
    const uint64_t uptr2 = reinterpret_cast<uint64_t>( ptr23 );
    i2 = uptr2 >> 32;
    i3 = uptr2 & 0x00000000ffffffff;

    const uint64_t uptr3 = reinterpret_cast<uint64_t>( ptr45 );
    i4 = uptr3 >> 32;
    i5 = uptr3 & 0x00000000ffffffff;

    const uint64_t uptr4 = reinterpret_cast<uint64_t>( ptr67 );
    i6 = uptr4 >> 32;
    i7 = uptr4 & 0x00000000ffffffff;
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD01()
  { 
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD23()
  { 
    const uint32_t u2 = optixGetPayload_2();
    const uint32_t u3 = optixGetPayload_3();
    return reinterpret_cast<T*>( unpackPointer( u2, u3 ) );
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD45()
  { 
    const uint32_t u4 = optixGetPayload_4();
    const uint32_t u5 = optixGetPayload_5();
    return reinterpret_cast<T*>( unpackPointer( u4, u5 ) );
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD67()
  { 
    const uint32_t u6 = optixGetPayload_6();
    const uint32_t u7 = optixGetPayload_7();
    return reinterpret_cast<T*>( unpackPointer( u6, u7 ) );
  }


  // for RANDOM THINGS =============================================
  template<unsigned int N>
  static __host__ __device__ __inline__ unsigned int tea( unsigned int val0, unsigned int val1 )
  {
    unsigned int v0 = val0;
    unsigned int v1 = val1;
    unsigned int s0 = 0;

    for( unsigned int n = 0; n < N; n++ )
    {
      s0 += 0x9e3779b9;
      v0 += ((v1<<4)+0xa341316c)^(v1+s0)^((v1>>5)+0xc8013ea4);
      v1 += ((v0<<4)+0xad90777d)^(v0+s0)^((v0>>5)+0x7e95761e);
    }

    return v0;
  }

  // Generate random unsigned int in [0, 2^24)
  static __host__ __device__ __inline__ unsigned int lcg(unsigned int &prev)
  {
    const unsigned int LCG_A = 1664525u;
    const unsigned int LCG_C = 1013904223u;
    prev = (LCG_A * prev + LCG_C);
    return prev & 0x00FFFFFF;
  }

    // Generate random float in [0, 1)
  static __host__ __device__ __inline__ float rnd(unsigned int &prev)
  {
    return ((float) lcg(prev) / (float) 0x01000000);
  }
  
  // generate random float in [min, max)
  static __host__ __device__ __inline__ float rnd(unsigned int &prev, float min, float max)
  {
    return min + (max-min)*rnd(prev);
  }

  static __host__ __device__ __inline__ float clamp(float x, float min, float max)
  {
    if (x < min) return min;
    if (x > max) return max;
    return x;
  }

  static __host__ __device__ __inline__ bool near_zero(vec3f v)
  {
    const float s = 1e-5;
    return (fabsf(v.x) < s) && (fabsf(v.y) < s) && (fabsf(v.z) < s);
  }

  static __host__ __device__ __inline__ float vec_length(vec3f v)
  {
    return sqrtf(v.x*v.x + v.y*v.y + v.z*v.z);
  }

  static __host__ __device__ __inline__ vec3f unit_vector(vec3f v)
  {
    return v / vec_length(v);
  }

  static __host__ __device__ __inline__ vec3f reflect(vec3f v, vec3f n)
  {
    return v - 2*dot(v,n)*n;
  }


  //------------------------------------------------------------------------------
  // closest hit and anyhit programs for radiance-type rays.
  //
  // Note eventually we will have to create one pair of those for each
  // ray type and each geometry type we want to render; but this
  // simple example doesn't use any actual geometries yet, so we only
  // create a single, dummy, set of them (we do have to have at least
  // one group of them to set up the SBT)
  //------------------------------------------------------------------------------
  
  extern "C" __global__ void __closesthit__radiance()
  {
    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // compute normal:
    const int   primID = optixGetPrimitiveIndex();
    const vec3i index  = sbtData.index[primID];
    const vec3f &A     = sbtData.vertex[index.x];
    const vec3f &B     = sbtData.vertex[index.y];
    const vec3f &C     = sbtData.vertex[index.z];
    const vec3f Ng     = -1.0f * normalize(cross(B-A,C-A));
    // const vec3f Ng     = normalize(cross(B-A,C-A));

    // const vec3f hit_point = 
    const vec3f rayOrig = optixGetWorldRayOrigin();
    const vec3f rayDir = optixGetWorldRayDirection();

    const float D = -1.0f*Ng.x*A.x - Ng.y*A.y - Ng.z*A.z;

    const float denom = dot(rayDir, Ng);
    const float cosDN  = 0.2f + .8f*fabsf(denom);
    const float t = -1.0f * (dot(Ng, rayOrig) + D) / denom;

    // location of the hit 
    const vec3f p_hit = rayOrig + rayDir * t;
    vec3f &prd_hit_p = *(vec3f*)getPRD23<vec3f>();
    prd_hit_p = p_hit + (Ng * 0.001f);

    const vec3f material = sbtData.material;

    // find the direction of the bounced ray
    // eventually this needs to be based on the sbtdata, which will carry the surface type. 
    
    // relay surface type back to raygen, because I want to use the random seed i started there.
    // i don't know how to ensure I get a random number here within closest hit. 
    // I probably need to pass &prev along with the 
    vec3f &prd_new_ray_dir = *(vec3f*)getPRD45<vec3f>();
    if (material.x == 0.0) 
    {
      // ----------- Scatter --------------
      unsigned int &prd_seed = *(unsigned int*)getPRD67<unsigned int>();
      
      float rx = 0.0;
      float ry = 0.0;
      float rz = 0.0;
      while (true)
      {
        rx = rnd( prd_seed, -1.f, 1.f);
        ry = rnd( prd_seed, -1.f, 1.f);
        rz = rnd( prd_seed, -1.f, 1.f);
        if (sqrtf(rx*rx + ry*ry + rz*rz) >= 1) continue;
        break;
      }
      // const vec3f new_ray_dir = normalize(vec3f(rx, ry, rz) + Ng);
      vec3f scatter_direction = vec3f(rx, ry, rz) + Ng;
      if (near_zero(scatter_direction))
        scatter_direction = Ng;
      prd_new_ray_dir = scatter_direction;
      // -----------------------------------
    } 
    else if (material.x == 1.0)
    {
      // ------------- Reflection ----------
      vec3f reflected = reflect(unit_vector(rayDir), Ng);
      prd_new_ray_dir = reflected;
      // -----------------------------------
    } 
    else if (material.x == 2.0)
    {
      //------------- Fuzzed --------------
      vec3f reflected = reflect(unit_vector(rayDir), Ng);
      float fuzz = material.y;
      unsigned int &prd_seed = *(unsigned int*)getPRD67<unsigned int>();
      
      float rx = 0.0;
      float ry = 0.0;
      float rz = 0.0;
      while (true)
      {
        rx = rnd( prd_seed, -1.f, 1.f);
        ry = rnd( prd_seed, -1.f, 1.f);
        rz = rnd( prd_seed, -1.f, 1.f);
        if (sqrtf(rx*rx + ry*ry + rz*rz) >= 1) continue;
        break;
      }
      vec3f fuzzed = reflected + (vec3f(rx, ry, rz) * fuzz);
      if (near_zero(fuzzed))
        fuzzed = reflected; 
      prd_new_ray_dir = fuzzed;
      //-----------------------------------
    }
    else if (material.x == 3.0)
    {
      //------------- light source --------------
      prd_hit_p = vec3f(999999.99f,999999.99f,999999.99f);
      //-----------------------------------
    }

    vec3f &prd = *(vec3f*)getPRD01<vec3f>();
    // prd = sbtData.color;
    prd = cosDN * sbtData.color;
    
  }
  
  extern "C" __global__ void __anyhit__radiance()
  { /*! for this simple example, this will remain empty */ }


  
  //------------------------------------------------------------------------------
  // miss program that gets called for any ray that did not have a
  // valid intersection
  //
  // as with the anyhit/closest hit programs, in this example we only
  // need to have _some_ dummy function to set up a valid SBT
  // ------------------------------------------------------------------------------
  
  extern "C" __global__ void __miss__radiance()
  {
    vec3f &prd = *(vec3f*)getPRD01<vec3f>();
    vec3f &prd_hit_p = *(vec3f*)getPRD23<vec3f>();
    // fake value for hit point, so I know to end the ray bouncing
    prd_hit_p = vec3f(999999.99f,999999.99f,999999.99f);

    // rayDir should be normalized becasue that happens in raygen__renderFrame()
    const vec3f rayDir = optixGetWorldRayDirection();
    // const vec3f n_rayDir = normalize(rayDir);

    const float t = 0.5*(rayDir.y + 1.0);
    const float tin = 1.0 - t;
    //(1.0-t)*vec3f(1.f) +
    //sky that fades to white
    prd = tin*vec3f(1.f,1.f,1.f) + t*vec3f(0.5f,0.7f,1.0f);
    // sky that's solid light blue
    // prd = vec3f(0.5f,0.7f,1.0f);
    // background black
    // prd = vec3f(0.f);
  }

  //------------------------------------------------------------------------------
  // ray gen program - the actual rendering happens in here
  //------------------------------------------------------------------------------
  extern "C" __global__ void __raygen__renderFrame()
  {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const uint3 dim = optixGetLaunchDimensions();

    const auto &camera = optixLaunchParams.camera;
    const uint32_t fbIndex = ix+iy*optixLaunchParams.frame.size.x;

    unsigned int seed = tea<4>( iy * dim.x + ix, 12346789 + fbIndex );
    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    vec3f rayColorPRD = vec3f(0.f);
    vec3f ray_hit_p = vec3f(0.f);
    vec3f ray_dir_new = vec3f(0.f);
    // the values we store the PRD pointer in:
    uint32_t u0, u1, u2, u3, u4, u5, u6, u7;
    packPointer( &rayColorPRD, &ray_hit_p, &ray_dir_new, &seed, u0, u1, u2, u3, u4, u5, u6, u7);

    // normalized screen plane position, in [0,1]^2
    vec2f screen = vec2f(0.f);
    
    const int samples_per_pixel = 20;

    float r = 0.0;
    float g = 0.0;
    float b = 0.0;

    for (int s = 0; s < samples_per_pixel; s++)
    {
      float rx = rnd( seed );
      float ry = rnd( seed );

      screen = vec2f(rx+ix+.5f,ry+iy+.5f)
                       / vec2f(optixLaunchParams.frame.size);
      // screen = vec2f(ix+.5f,iy+.5f)
      //                  / vec2f(optixLaunchParams.frame.size);

      ray_dir_new = normalize(camera.direction
                             + (screen.x - 0.5f) * camera.horizontal
                             + (screen.y - 0.5f) * camera.vertical);
      
      ray_hit_p = camera.position;

      int max_ray_depth = 50;
      int bounces = 0;

      float temp_r = 1.0;
      float temp_g = 1.0;
      float temp_b = 1.0;

      while (bounces < max_ray_depth)
      {
      // // for (int depth=0; depth<max_ray_depth; depth++)
      // // {
        // int num_bounces = 0;
        optixTrace(optixLaunchParams.traversable,
                  ray_hit_p,
                  ray_dir_new,
                  0.f,    // tmin
                  1e20f,  // tmax
                  0.0f,   // rayTime
                  OptixVisibilityMask( 255 ),
                  OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                  SURFACE_RAY_TYPE,             // SBT offset
                  RAY_TYPE_COUNT,               // SBT stride
                  SURFACE_RAY_TYPE,             // missSBTIndex 
                  u0, u1, u2, u3, u4, u5, u6, u7 );
        temp_r *= rayColorPRD.x;
        temp_g *= rayColorPRD.y;
        temp_b *= rayColorPRD.z;
        if (ray_hit_p.x == 999999.99f)
          break;
        bounces++;
      }
      if (bounces == max_ray_depth && max_ray_depth != 1) {
        temp_r *= 0;
        temp_g *= 0;
        temp_b *= 0;
      }
      r += temp_r;
      g += temp_g;
      b += temp_b;
    }
    // scale by the number of samples, then scale to 256
    const int r_out = int(clamp(sqrtf(r/samples_per_pixel), 0.0f, 0.999f) * 255.99f);
    const int g_out = int(clamp(sqrtf(g/samples_per_pixel), 0.0f, 0.999f) * 255.99f);
    const int b_out = int(clamp(sqrtf(b/samples_per_pixel), 0.0f, 0.999f) * 255.99f);

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000
      | (r_out<<0) | (g_out<<8) | (b_out<<16);

    // and write to frame buffer ...
    
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
  }
  
} // ::osc
