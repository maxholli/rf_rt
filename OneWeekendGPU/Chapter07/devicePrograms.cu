#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>

#include "LaunchParams.h"

using namespace osc;

namespace osc {
  
  /*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
  extern "C" __constant__ LaunchParams optixLaunchParams;

  // for this simple example, we have a single ray type
  enum { SURFACE_RAY_TYPE=0, RAY_TYPE_COUNT };
  
  static __forceinline__ __device__
  void *unpackPointer( uint32_t i0, uint32_t i1 )
  {
    const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
    void*           ptr = reinterpret_cast<void*>( uptr ); 
    return ptr;
  }

  static __forceinline__ __device__
  void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
  {
    const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
    i0 = uptr >> 32;
    i1 = uptr & 0x00000000ffffffff;
  }

  template<typename T>
  static __forceinline__ __device__ T *getPRD()
  { 
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
  }


  template<unsigned int N>
  static __host__ __device__ __inline__ unsigned int tea( unsigned int val0, unsigned int val1 )
  {
    unsigned int v0 = val0;
    unsigned int v1 = val1;
    unsigned int s0 = 0;

    for( unsigned int n = 0; n < N; n++ )
    {
      s0 += 0x9e3779b9;
      v0 += ((v1<<4)+0xa341316c)^(v1+s0)^((v1>>5)+0xc8013ea4);
      v1 += ((v0<<4)+0xad90777d)^(v0+s0)^((v0>>5)+0x7e95761e);
    }

    return v0;
  }

  // Generate random unsigned int in [0, 2^24)
  static __host__ __device__ __inline__ unsigned int lcg(unsigned int &prev)
  {
    const unsigned int LCG_A = 1664525u;
    const unsigned int LCG_C = 1013904223u;
    prev = (LCG_A * prev + LCG_C);
    return prev & 0x00FFFFFF;
  }

    // Generate random float in [0, 1)
  static __host__ __device__ __inline__ float rnd(unsigned int &prev)
  {
    return ((float) lcg(prev) / (float) 0x01000000);
  }
  
  //------------------------------------------------------------------------------
  // closest hit and anyhit programs for radiance-type rays.
  //
  // Note eventually we will have to create one pair of those for each
  // ray type and each geometry type we want to render; but this
  // simple example doesn't use any actual geometries yet, so we only
  // create a single, dummy, set of them (we do have to have at least
  // one group of them to set up the SBT)
  //------------------------------------------------------------------------------
  
  extern "C" __global__ void __closesthit__radiance()
  {
    const TriangleMeshSBTData &sbtData
      = *(const TriangleMeshSBTData*)optixGetSbtDataPointer();

    // compute normal:
    const int   primID = optixGetPrimitiveIndex();
    const vec3i index  = sbtData.index[primID];
    const vec3f &A     = sbtData.vertex[index.x];
    const vec3f &B     = sbtData.vertex[index.y];
    const vec3f &C     = sbtData.vertex[index.z];
    const vec3f Ng     = normalize(cross(B-A,C-A));

    const vec3f rayDir = optixGetWorldRayDirection();
    const float cosDN  = 0.2f + .8f*fabsf(dot(rayDir,Ng));
    vec3f &prd = *(vec3f*)getPRD<vec3f>();
    prd = cosDN * sbtData.color;
  }
  
  extern "C" __global__ void __anyhit__radiance()
  { /*! for this simple example, this will remain empty */ }


  
  //------------------------------------------------------------------------------
  // miss program that gets called for any ray that did not have a
  // valid intersection
  //
  // as with the anyhit/closest hit programs, in this example we only
  // need to have _some_ dummy function to set up a valid SBT
  // ------------------------------------------------------------------------------
  
  extern "C" __global__ void __miss__radiance()
  {
    vec3f &prd = *(vec3f*)getPRD<vec3f>();

    // rayDir should be normalized becasue that happens in raygen__renderFrame()
    const vec3f rayDir = optixGetWorldRayDirection();
    // const vec3f n_rayDir = normalize(rayDir);

    const float t = 0.5*(rayDir.y + 1.0);
    const float tin = 1.0 - t;
    //(1.0-t)*vec3f(1.f) +
    prd = tin*vec3f(1.f,1.f,1.f) + t*vec3f(0.5f,0.7f,1.0f);
  }

  //------------------------------------------------------------------------------
  // ray gen program - the actual rendering happens in here
  //------------------------------------------------------------------------------
  extern "C" __global__ void __raygen__renderFrame()
  {
    // compute a test pattern based on pixel ID
    const int ix = optixGetLaunchIndex().x;
    const int iy = optixGetLaunchIndex().y;
    const uint3 dim = optixGetLaunchDimensions();

    const auto &camera = optixLaunchParams.camera;
    const uint32_t fbIndex = ix+iy*optixLaunchParams.frame.size.x;

    unsigned int seed = tea<4>( iy * dim.x + ix, 12346789 + fbIndex );
    // our per-ray data for this example. what we initialize it to
    // won't matter, since this value will be overwritten by either
    // the miss or hit program, anyway
    vec3f rayColorPRD = vec3f(0.f);
    // the values we store the PRD pointer in:
    uint32_t u0, u1;
    packPointer( &rayColorPRD, u0, u1 );

    // normalized screen plane position, in [0,1]^2
    vec2f screen = vec2f(0.f);
    
    // generate ray direction
    vec3f rayDir = vec3f(0.f);
    
    const int samples_per_pixel = 100;

    float r = 0.0;
    float g = 0.0;
    float b = 0.0;

    for (int s = 0; s < samples_per_pixel; s++)
    {
      float rx = rnd( seed );
      float ry = rnd( seed );

      screen = vec2f(rx+ix+.5f,ry+iy+.5f)
                       / vec2f(optixLaunchParams.frame.size);

      rayDir = normalize(camera.direction
                             + (screen.x - 0.5f) * camera.horizontal
                             + (screen.y - 0.5f) * camera.vertical);
      optixTrace(optixLaunchParams.traversable,
                camera.position,
                rayDir,
                0.f,    // tmin
                1e20f,  // tmax
                0.0f,   // rayTime
                OptixVisibilityMask( 255 ),
                OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                SURFACE_RAY_TYPE,             // SBT offset
                RAY_TYPE_COUNT,               // SBT stride
                SURFACE_RAY_TYPE,             // missSBTIndex 
                u0, u1 );
      r += rayColorPRD.x;
      g += rayColorPRD.y;
      b += rayColorPRD.z; 
    }
    // scale by the number of samples, then scale to 256
    const int r_out = int(r/samples_per_pixel * 255.99f);
    const int g_out = int(g/samples_per_pixel * 255.99f);
    const int b_out = int(b/samples_per_pixel * 255.99f);

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000
      | (r_out<<0) | (g_out<<8) | (b_out<<16);

    // and write to frame buffer ...
    
    optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
  }
  
} // ::osc
